#include "hip/hip_runtime.h"
/**
 * Copyright (C) Mellanox Technologies Ltd. 2022.  ALL RIGHTS RESERVED.
 *
 * See file LICENSE for terms.
 */

#ifdef __cplusplus
extern "C" {
#endif

#include "../ec_cuda.h"
#include "utils/ucc_math.h"
#include <inttypes.h>

#ifdef __cplusplus
}
#endif

#define align_pow2(_n, _p) ((_n) & ((_p) - 1))

__global__ void executor_start(ucc_ec_cuda_executor_state_t *state,
                               int *cidx)
{
    *cidx  = 0;
    *state = UCC_EC_CUDA_EXECUTOR_STARTED;
}

__global__ void executor_shutdown_ack(ucc_ec_cuda_executor_state_t *state)
{
    *state = UCC_EC_CUDA_EXECUTOR_SHUTDOWN_ACK;
}

template <typename T>
__device__ void executor_copy(T* __restrict__ d, T* __restrict__ s,
                              size_t count)
{
    size_t start = threadIdx.x;
    const size_t step  = blockDim.x;

    for (size_t i = start; i < count; i+=step) {
        d[i] = s[i];
    }
}

template <typename T>
__device__ void executor_copy_aligned(T* __restrict__ d, T* __restrict__ s,
                                      size_t count)
{
    size_t idx = threadIdx.x;
    const size_t step  = blockDim.x;
    const int n = count / sizeof(T);
    const int num_iter = n / step + ((idx < n % step) ? 1 : 0);
    char1 *s1 = (char1*)s;
    char1 *d1 = (char1*)d;

#pragma unroll
    for(int i = 0; i < num_iter; i++) {
        d[i * step + idx] = s[i * step + idx];
    }

    if (idx < count % sizeof(T)) {
        d1[count - idx - 1] = s1[count - idx - 1];
    }
}

__device__ inline void add_float4(float4 &d, const float4 &x, const float4 &y)
{
    d.x = x.x + y.x;
    d.y = x.y + y.y;
    d.z = x.z + y.z;
    d.w = x.w + y.w;
}

__device__ void executor_reduce_float(const float *s1, const float *s2,
                                      float *d, size_t count)
{
    const float4 *s14      = (const float4*)s1;
    const float4 *s24      = (const float4*)s2;
    float4       *d4       = (float4*)d;
    const size_t  idx      = threadIdx.x;
    const size_t  step     = blockDim.x;
    const int     n        = count / 4;
    const int     num_iter = n / step + ((idx < n % step) ? 1 : 0);

    for(int i = 0; i < num_iter; i++) {
        add_float4(d4[i * step + idx], s14[i * step + idx],
                   s24[i * step + idx]);
    }
    if (idx < count % 4) {
        d[count - idx - 1] = s1[count - idx - 1] + s2[count - idx - 1];
    }
}

template <typename T>
__device__ void executor_reduce(const T* __restrict__ s1,
                                const T* __restrict__ s2,
                                T* __restrict__ d, size_t count)
{
    const size_t step  = blockDim.x;
    const size_t start = threadIdx.x;

    for (size_t i = start; i < count; i+=step) {
        d[i] = s1[i] + s2[i];
    }
}

template <typename T>
__device__ void executor_reduce_multi(const T* __restrict__ s1,
                                      const T* __restrict__ s2,
                                      T* __restrict__ d, size_t count,
                                      size_t size, size_t stride)
{
    const size_t step  = blockDim.x;
    const size_t start = threadIdx.x;
    const size_t ld    = stride / sizeof(T);

    for (size_t i = start; i < count; i+=step) {
        d[i] = s1[i] + s2[i];
        for (size_t j = 1; j < size; j++) {
            d[i] = d[i] + s2[i + j*ld];
        }
    }
}

__device__ void executor_copy_multi(ucc_ee_executor_task_args_copy_multi_t args)
{
    const size_t     step     = blockDim.x;
    size_t           min_size = args.counts[0];
    size_t           idx      = threadIdx.x;
    __shared__ int4 *dsts[UCC_EE_EXECUTOR_NUM_COPY_BUFS];
    __shared__ int4 *srcs[UCC_EE_EXECUTOR_NUM_COPY_BUFS];
    bool             aligned;

    for (int i = 0; i < args.num_vectors; i++) {
        dsts[i] = (int4*)args.dst[i];
        srcs[i] = (int4*)args.src[i];
        aligned = !(align_pow2((intptr_t)srcs[i], 16) ||
                    align_pow2((intptr_t)dsts[i], 16));
        if (!aligned) {
            break;
        }
        if (args.counts[i] < min_size) {
            min_size = args.counts[i];
        }
    }

    if (!aligned || min_size < 16) {
        for (int i = 0; i < args.num_vectors; i++) {
            executor_copy((char*)args.dst[i],
                          (char*)args.src[i],
                          args.counts[i]);
        }
        return;
    }

    const int n        = min_size / sizeof(uint4);
    const int num_iter = n / step + ((threadIdx.x < n % step) ? 1 : 0);

    for (size_t i = 0; i < num_iter; i++) {
#pragma unroll
        for (int j = 0; j < args.num_vectors; j++) {
            dsts[j][idx] = srcs[j][idx];
        }
        idx += step;
    }

    const size_t left = min_size + min_size % sizeof(uint4);

    for (int i = 0; i < args.num_vectors; i++) {
        executor_copy((char*)args.dst[i] + left,
                      (char*)args.src[i] + left,
                      args.counts[i] - left);
    }
}


__global__ void executor_kernel(volatile ucc_ec_cuda_executor_t *eee,
                                int q_size)
{
    const uint32_t  worker_id   = blockIdx.x;
    const uint32_t  num_workers = gridDim.x;
    bool            is_master   = (threadIdx.x == 0) ? true: false;
    int cidx_local, pidx_local;
    volatile int *pidx, *cidx;
    ucc_ee_executor_task_t *tasks;
    __shared__ ucc_ee_executor_task_args_t args;
    __shared__ bool worker_done;

    if (is_master) {
        cidx_local = worker_id;
        pidx       = eee->dev_pidx;
        cidx       = eee->dev_cidx;
        tasks      = eee->dev_tasks;
    }

    worker_done = false;
    __syncthreads();
    while (1) {
        if (is_master) {
            while ((*cidx % num_workers) != worker_id);
            do {
                pidx_local = *pidx;
            } while (*cidx == pidx_local);
            (*cidx)++;
            worker_done = (pidx_local == -1);
            if (!worker_done) {
                args = tasks[cidx_local].args;
            }
        }
        __syncthreads();
        if (worker_done) {
            return;
        }
        switch (args.task_type) {
            bool aligned;
            case UCC_EE_EXECUTOR_TASK_TYPE_COPY:
                aligned = !(align_pow2((intptr_t)args.bufs[0], 16) ||
                            align_pow2((intptr_t)args.bufs[1], 16));
                if (aligned) {
                    executor_copy_aligned<uint4>((uint4*)args.bufs[0],
                                                 (uint4*)args.bufs[1],
                                                 args.count);

                } else {
                    executor_copy((char*)args.bufs[0],
                                  (char*)args.bufs[1],
                                   args.count);
                }
                break;
            case UCC_EE_EXECUTOR_TASK_TYPE_COPY_MULTI:
                executor_copy_multi(args.copy_multi);
                break;
            case UCC_EE_EXECUTOR_TASK_TYPE_REDUCE:
                aligned = !(align_pow2((intptr_t)args.bufs[0], 16) ||
                            align_pow2((intptr_t)args.bufs[1], 16) ||
                            align_pow2((intptr_t)args.bufs[2], 16));
                switch (args.dt)
                {
                case UCC_DT_FLOAT32:
                    if (aligned) {
                        executor_reduce_float((float*)args.bufs[1],
                                              (float*)args.bufs[2],
                                              (float*)args.bufs[0],
                                              args.count);
                    } else {
                        executor_reduce<float>((float*)args.bufs[1],
                                               (float*)args.bufs[2],
                                               (float*)args.bufs[0],
                                               args.count);
                    }
                    break;
                case UCC_DT_FLOAT64:
                    executor_reduce<double>((double*)args.bufs[1],
                                            (double*)args.bufs[2],
                                            (double*)args.bufs[0],
                                             args.count);
                    break;
                case UCC_DT_INT32:
                    executor_reduce<int32_t>((int32_t*)args.bufs[1],
                                             (int32_t*)args.bufs[2],
                                             (int32_t*)args.bufs[0],
                                              args.count);
                    break;

                default:
                    break;
                }
                break;
            case UCC_EE_EXECUTOR_TASK_TYPE_REDUCE_MULTI:
                switch(args.dt) {
                case UCC_DT_FLOAT32:
                    executor_reduce_multi<float>((float*)args.bufs[1],
                                                 (float*)args.bufs[2],
                                                 (float*)args.bufs[0],
                                                 args.count, args.size,
                                                 args.stride);
                    break;
                case UCC_DT_FLOAT64:
                    executor_reduce_multi<double>((double*)args.bufs[1],
                                                  (double*)args.bufs[2],
                                                  (double*)args.bufs[0],
                                                  args.count, args.size,
                                                  args.stride);
                    break;
                case UCC_DT_INT32:
                    executor_reduce_multi<int32_t>((int32_t*)args.bufs[1],
                                                   (int32_t*)args.bufs[2],
                                                   (int32_t*)args.bufs[0],
                                                   args.count, args.size,
                                                   args.stride);
                    break;
                }
                break;
            default: break;
        }
        __syncthreads();
        __threadfence_system();
        if (is_master) {
            tasks[cidx_local].status = UCC_OK;
            cidx_local = (cidx_local + num_workers) %q_size;
        }
    }
}

#ifdef __cplusplus
extern "C" {
#endif

ucc_status_t ucc_ec_cuda_persistent_kernel_start(ucc_ec_cuda_executor_t *eee)
{
    hipStream_t stream = (hipStream_t)eee->super.ee_context;
    int          nb     = EC_CUDA_CONFIG->exec_num_workers;
    int          nt     = EC_CUDA_CONFIG->exec_num_threads;
    int          q_size = EC_CUDA_CONFIG->exec_max_tasks;

    executor_start<<<1, 1, 0, stream>>>(eee->dev_state, eee->dev_cidx);
    executor_kernel<<<nb, nt, 0, stream>>>(eee, q_size);
    executor_shutdown_ack<<<1, 1, 0, stream>>>(eee->dev_state);
    CUDA_CHECK(hipGetLastError());

    return UCC_OK;
}

__global__ void kernel_copy_multi(ucc_ee_executor_task_args_copy_multi_t args)
{
    int     blocks_per_buf = gridDim.x / args.num_vectors;
    int     buf_id         = blockIdx.x / blocks_per_buf;
    char1  *src            = (char1*)args.src[buf_id];
    char1  *dst            = (char1*)args.dst[buf_id];
    size_t  cnt            = args.counts[buf_id];
    size_t  start          = threadIdx.x + (blockIdx.x % blocks_per_buf) * blockDim.x;
    size_t  step           = blockDim.x * blocks_per_buf;

    for (size_t i = start; i < cnt; i += step) {
        dst[i] = src[i];
    }
}

__global__ void kernel_copy_multi_aligned(ucc_ee_executor_task_args_copy_multi_t args)
{
    int    blocks_per_buf = gridDim.x / args.num_vectors;
    int    buf_id         = blockIdx.x / blocks_per_buf;
    int    idx            = threadIdx.x + (blockIdx.x % blocks_per_buf) * blockDim.x;
    int    step           = blockDim.x * blocks_per_buf;
    size_t n              = args.counts[buf_id] / sizeof(uint4);
    size_t num_iter       = n / step + ((idx < n % step) ? 1 : 0);
    uint4 *src            = (uint4*)args.src[buf_id];
    uint4 *dst            = (uint4*)args.dst[buf_id];

    for(size_t i = 0; i < num_iter; i++) {
        dst[i * step + idx] = src[i * step + idx];
    }

    if (idx < (args.counts[buf_id] % sizeof(uint4))) {
        ((char*)args.dst[buf_id])[args.counts[buf_id] - idx - 1] =
            ((char*)args.src[buf_id])[args.counts[buf_id] - idx - 1];
    }
}

ucc_status_t ucc_ec_cuda_copy_multi_kernel(const ucc_ee_executor_task_args_t *args,
                                           hipStream_t stream)
{
    int nt = 1024;
    int nb = args->copy_multi.num_vectors * 4;
    int aligned = 1;

    for (int i = 0; i < args->copy_multi.num_vectors; i++) {
        if (align_pow2((intptr_t)args->copy_multi.src[i], 16) ||
            align_pow2((intptr_t)args->copy_multi.dst[i], 16)) {
            aligned = 0;
            break;
        }
    }

    if (aligned) {
        kernel_copy_multi_aligned<<<nb, nt, 0, stream>>>(args->copy_multi);
    } else {
        kernel_copy_multi<<<nb, nt, 0, stream>>>(args->copy_multi);
    }
    CUDA_CHECK(hipGetLastError());
    return UCC_OK;
}

#ifdef __cplusplus
}
#endif
