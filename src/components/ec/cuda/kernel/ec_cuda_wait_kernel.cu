#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2021, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See file LICENSE for terms.
 */

#ifdef __cplusplus
extern "C" {
#endif

#include "../ec_cuda.h"
#ifdef __cplusplus
}
#endif

__global__ void wait_kernel(volatile uint32_t *status) {
    ucc_status_t st;
    *status = UCC_EC_CUDA_TASK_STARTED;
    do {
        st = (ucc_status_t)*status;
    } while(st != UCC_EC_CUDA_TASK_COMPLETED);
    *status = UCC_EC_CUDA_TASK_COMPLETED_ACK;
    return;
}

__global__ void wait_kernel_nb(volatile uint32_t *status) {
    *status = UCC_EC_CUDA_TASK_COMPLETED_ACK;
    return;
}

#ifdef __cplusplus
extern "C" {
#endif

ucc_status_t ucc_ec_cuda_post_kernel_stream_task(uint32_t *status,
                                                 int blocking_wait,
                                                 hipStream_t stream)
{
    if (blocking_wait) {
        wait_kernel<<<1, 1, 0, stream>>>(status);
    } else {
        wait_kernel_nb<<<1, 1, 0, stream>>>(status);
    }
    CUDA_CHECK(hipGetLastError());
    return UCC_OK;
}

#ifdef __cplusplus
}
#endif
